﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <fstream>
#include <iostream>

__global__ void manipulateArray(long* indexArray, long from, long amount)
{
    long i = threadIdx.x;

    indexArray[from + i] += amount;
}

struct inputQueryRow
{
    long rangeMin, rangeMax, amount;
};

struct input
{
    long indexArrayCount, queryCount;
    struct inputQueryRow* inputQueryRow;
};



long maxSum(struct input* inputData)
{
    long* indexArray = (long*)malloc(inputData->indexArrayCount * sizeof(long));
    memset(indexArray, 0, inputData->indexArrayCount * sizeof(long));
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);


    //! create memory space for indexArray in GPU.
    long* devGPUMemory = 0;
    cudaStatus = hipMalloc( (void**)&devGPUMemory, inputData->indexArrayCount * sizeof(long) );
    cudaStatus = hipMemcpy(devGPUMemory, indexArray, inputData->indexArrayCount * sizeof(long), hipMemcpyHostToDevice);

    for (long i = 0; i < inputData->queryCount; i++)
    {
        long rangeMin = inputData->inputQueryRow[i].rangeMin;
        long rangeMax = inputData->inputQueryRow[i].rangeMax;
        long amount = inputData->inputQueryRow[i].amount;


        int curThreads = 0;
        int curCursor = 0;

        int rangeSize = rangeMax - rangeMin;

        for (int j = rangeMin - 1; j < rangeMax; j += 1024)
        {
            curCursor = j;
            curThreads = (j + 1024) > rangeMax ? rangeMax - j : 1024;

            //std::cout << "curCursor: " << curCursor << " curThreads: " << curThreads << std::endl;

            manipulateArray <<<1, curThreads >>> (devGPUMemory, curCursor, amount);
        }
    }


    cudaStatus = hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(indexArray, devGPUMemory, inputData->indexArrayCount * sizeof(long), hipMemcpyDeviceToHost);

    long max = 0;
    for (long i = 0; i < inputData->indexArrayCount; i++)
    {
        //std::cout << indexArray[i] << " ";
        
        if (indexArray[i] > max) max = indexArray[i];
    }
    std::cout << std::endl;

    hipFree(devGPUMemory);

    return max;
}

int main()
{
    std::fstream fin("input.txt");

    std::cout << "Matrix Manipulation using GPU." << std::endl;

    struct input newInput {};

    fin >> newInput.indexArrayCount >> newInput.queryCount;

    newInput.inputQueryRow = (struct inputQueryRow*)malloc(newInput.queryCount * sizeof(struct inputQueryRow));

    for (long i = 0; i < newInput.queryCount; i++)
        fin >> newInput.inputQueryRow[i].rangeMin >> newInput.inputQueryRow[i].rangeMax >> newInput.inputQueryRow[i].amount;

    long max = maxSum(&newInput);

    std::cout << "Max Sum: " << max << std::endl;

    return 0;
}